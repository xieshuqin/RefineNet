#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "weighted_sigmoid_cross_entropy_loss_op.h"

namespace caffe2 {

namespace {
__global__ void ElementwiseMaxKernel(const int n, float* data, const float a) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    data[index] = (data[index] > a) ? data[index] : a;
  }
}

__global__ void WeightedSigmoidCrossEntropyLossKernel(
    const int n,
    const float pos_weight,
    const float neg_weight,
    const float* logits,
    const int* targets,
    float* losses,
    float* counts) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    if (targets[index] < 0. ) {
      losses[index] = 0.;
      counts[index] = 0.;
    } else {
      float c1 = targets[index];
      float c2 = 1. - targets[index];

      // p = 1. / 1. + expf(-x)
      float p = 1. / (1. + expf(-logits[index]));

      // (1 - p)**gamma * log(p) where
      float term1 = pos_weight * logf(max(p, FLT_MIN));
      // p**gamma * log(1 - p)
      float term2 = neg_weight *
          (-1. * logits[index] * (logits[index] >= 0) - logf(1. + 
            expf(logits[index] - 2. * logits[index] * (logits[index] >= 0))));

      losses[index] = - (c1 * term1 + c2 * term2); 
      counts[index] = 1.;
    }
  }
}

__global__ void WeightedSigmoidCrossEntropyLossGradientKernel(
    const int n,
    const float pos_weight,
    const float neg_weight,
    const float* logits,
    const int* targets,
    float* d_logits,
    float* counts) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    if (targets[index] < 0.) {
      d_logits[index] = 0.;
      counts[index] = 0.;
    } else {
      float c1 = targets[index];
      float c2 = (1. - targets[index]);
      float p = 1. / (1. + expf(-logits[index]));

      // (1-p)**g * (1 - p - g*p*log(p))
      float term1 = pos_weight * (1. - p);
      // (p**g) * (g*(1-p)*log(1-p) - p)
      float term2 = neg_weight * ( - p);

      d_logits[index] = -(c1 * term1 + c2 * term2);
      counts[index] = 1.;
    }
  }
}
} // namespace

template <>
bool WeightedSigmoidCrossEntropyLossOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& T = Input(1);
  auto* avg_loss = Output(0);

  CAFFE_ENFORCE(
      X.size() == T.size(),
      "Logit and target must have the same size",
      "(",
      X.size(),
      " vs. ",
      T.size(),
      ")");
  avg_loss->Resize(vector<TIndex>());
  counts_.ResizeLike(X);
  losses_.ResizeLike(X);
  normalizer_.Resize(vector<TIndex>());
  WeightedSigmoidCrossEntropyLossKernel<<<
      CAFFE_GET_BLOCKS(X.size()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.size(),
      pos_weight_,
      neg_weight_,
      X.data<float>(),
      T.data<int>(),
      losses_.mutable_data<float>(),
      counts_.mutable_data<float>());
  float* avg_loss_data = avg_loss->mutable_data<float>();
  math::Sum<float, HIPContext>(
      losses_.size(), losses_.data<float>(), avg_loss_data, &context_);
  if (normalize_) {
    float* normalizer_data = normalizer_.mutable_data<float>();
    math::Sum<float, HIPContext>(
        counts_.size(), counts_.data<float>(), normalizer_data, &context_);
    // Prevent division by zero is all counts are zero
    ElementwiseMaxKernel<<<
        CAFFE_GET_BLOCKS(normalizer_.size()),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(normalizer_.size(), normalizer_data, 1e-5);
    math::Div<float, HIPContext>(
        1, avg_loss_data, normalizer_data, avg_loss_data, &context_);
  }
  math::Scale<float, HIPContext>(
      1, scale_, avg_loss_data, avg_loss_data, &context_);

  return true;
}

template <>
bool WeightedSigmoidCrossEntropyLossGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& T = Input(1);
  auto& d_avg_loss = Input(2);
  auto* dX = Output(0);

  dX->ResizeLike(X);
  counts_.ResizeLike(X);
  normalizer_.Resize(vector<TIndex>());
  WeightedSigmoidCrossEntropyLossGradientKernel<<<
      CAFFE_GET_BLOCKS(X.size()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.size(),
      pos_weight_,
      neg_weight_,
      X.data<float>(),
      T.data<int>(),
      dX->mutable_data<float>(),
      counts_.mutable_data<float>());
  if (normalize_) {
    float* normalizer_data = normalizer_.mutable_data<float>();
    math::Sum<float, HIPContext>(
        counts_.size(), counts_.data<float>(), normalizer_data, &context_);
    // Prevent division by zero is all counts are zero
    ElementwiseMaxKernel<<<
        CAFFE_GET_BLOCKS(normalizer_.size()),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(normalizer_.size(), normalizer_data, 1e-5);
    math::Div<float, HIPContext>(
        1,
        d_avg_loss.data<float>(),
        normalizer_data,
        normalizer_data,
        &context_);
    math::Scale<float, HIPContext>(
        1, scale_, normalizer_data, normalizer_data, &context_);
    math::Scale<float, HIPContext>(
        dX->size(),
        normalizer_data,
        dX->data<float>(),
        dX->mutable_data<float>(),
        &context_);
  } else {
    math::Scale<float, HIPContext>(
        dX->size(),
        scale_,
        dX->data<float>(),
        dX->mutable_data<float>(),
        &context_);
    math::Scale<float, HIPContext>(
        dX->size(),
        d_avg_loss.data<float>(),
        dX->data<float>(),
        dX->mutable_data<float>(),
        &context_);
  }
  return true;
}

REGISTER_HIP_OPERATOR(
    WeightedSigmoidCrossEntropyLoss,
    WeightedSigmoidCrossEntropyLossOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    WeightedSigmoidCrossEntropyLossGradient,
    WeightedSigmoidCrossEntropyLossGradientOp<float, HIPContext>);
} // namespace caffe2
